#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2020 MACNICA Inc.

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

// System includes
#include <stdio.h>
#include <assert.h>
#include <float.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

__global__ void maxPooling( \
    float *src, unsigned int srcHeight, unsigned int srcWidth, \
    float *dst,  unsigned int dstHeight, unsigned int dstWidth, \
    uint2 window, uint2 stride)
{
    unsigned int dstX = threadIdx.x + blockIdx.x * blockDim.x;
    if (dstX >= dstWidth) {
        return;
    }
    unsigned int dstY = threadIdx.y + blockIdx.y * blockDim.y;
    if (dstY >= dstHeight) {
      return;
    }
    unsigned int dstIdx = dstY * dstWidth + dstX;

    unsigned int srcX = dstX * stride.x;
    if (srcX >= srcWidth) {
        return;
    }
    unsigned int srcY = dstY * stride.y;
    if (srcY >= srcHeight) {
        return;
    }
    unsigned int srcIdx = srcY * srcWidth + srcX;

    float maxVal = -FLT_MAX;
    unsigned int idx;

    for (int h = 0;h < window.y;h++) {
        idx = srcIdx;
        for (int w = 0;w < window.x;w++) {
            maxVal = MAX(src[idx], maxVal);
            idx += 1;
        }
        srcIdx += srcWidth;
    }

    dst[dstIdx] = maxVal;
}

__global__ void avgPooling( \
    float *src, unsigned int srcHeight, unsigned int srcWidth, \
    float *dst,  unsigned int dstHeight, unsigned int dstWidth, \
    uint2 window, uint2 stride)
{
    unsigned int dstX = threadIdx.x + blockIdx.x * blockDim.x;
    if (dstX >= dstWidth) {
        return;
    }
    unsigned int dstY = threadIdx.y + blockIdx.y * blockDim.y;
    if (dstY >= dstHeight) {
      return;
    }
    unsigned int dstIdx = dstY * dstWidth + dstX;

    unsigned int srcX = dstX * stride.x;
    if (srcX >= srcWidth) {
        return;
    }
    unsigned int srcY = dstY * stride.y;
    if (srcY >= srcHeight) {
        return;
    }
    unsigned int srcIdx = srcY * srcWidth + srcX;

    float sum = 0.0;
    unsigned int idx;

    for (int h = 0;h < window.y;h++) {
        idx = srcIdx;
        for (int w = 0;w < window.x;w++) {
            sum += src[idx];
            idx += 1;
        }
        srcIdx += srcWidth;
    }

    dst[dstIdx] = sum / (window.y * window.x);
}

void cudaPooling(int type, \
    float *src, unsigned int srcHeight, int srcWidth, \
    float *dst, unsigned int dstHeight, int dstWidth, \
    int windowHeight, int windowWidth, \
    int strideHeight, int strideWidth, hipStream_t stream)
{
    dim3 block(32, 32);
    int grid_x = (dstWidth + block.x - 1) / block.x;
    int grid_y = (dstHeight + block.y - 1) / block.y;
    dim3 grid(grid_x, grid_y);
    uint2 window = make_uint2(windowWidth, windowHeight);
    uint2 stride = make_uint2(strideWidth, strideHeight);

    if (type) {
        maxPooling<<<grid, block, 0, stream>>>( \
            src, srcHeight, srcWidth, dst, dstHeight, dstWidth, window, stride);
    }
    else {
        avgPooling<<<grid, block, 0, stream>>>( \
            src, srcHeight, srcWidth, dst, dstHeight, dstWidth, window, stride);
    }
}
